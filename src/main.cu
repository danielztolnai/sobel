#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#include <stdint.h>
#include <hip/hip_runtime.h>
#include "image.h"

#define RUNS 200

#define TBW 32 // ThreadBlock width
#define TBH  8 // ThreadBlock height
#define TBWE (TBW+2)
#define TBHE (TBH+2)

__constant__ float xFilterSobel[3][3] = {
    {-1.0f,  0.0f,  1.0f},
    {-2.0f,  0.0f,  2.0f},
    {-1.0f,  0.0f,  1.0f}
};

__constant__ float yFilterSobel[3][3] = {
    {-1.0f, -2.0f, -1.0f},
    { 0.0f,  0.0f,  0.0f},
    { 1.0f,  2.0f,  1.0f}
};

__global__ void filterSobelCuda(unsigned char* imageIn, unsigned char* imageOut, int width)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int id  = threadIdx.y * blockDim.x + threadIdx.x;
    int loadAddress = (blockIdx.y*blockDim.y)*(width+2) + blockIdx.x*blockDim.x;
    float xValue=0.0f, yValue=0.0f;

    __shared__ unsigned char window[TBHE][TBWE];
    if(id < TBWE*TBHE/2) {
        window[(id/TBWE)       ][id%TBWE] = imageIn[loadAddress + ((id/TBWE)       )*(width+2) + id%TBWE];
        window[(id/TBWE)+TBHE/2][id%TBWE] = imageIn[loadAddress + ((id/TBWE)+TBHE/2)*(width+2) + id%TBWE];
    }
    __syncthreads();

    #pragma unroll 3
    for(int mrow=0; mrow<3; ++mrow) {
        #pragma unroll 3
        for(int mcol=0; mcol<3; ++mcol) {
            xValue += (float)(window[threadIdx.y + mrow][threadIdx.x + mcol]) * xFilterSobel[mrow][mcol];
            yValue += (float)(window[threadIdx.y + mrow][threadIdx.x + mcol]) * yFilterSobel[mrow][mcol];
        }
    }
    imageOut[row*width + col] = (unsigned char)fmaxf(fminf(sqrtf(xValue*xValue + yValue*yValue),255.0f),0.0f);
}

int main(int argc, char **argv)
{
    if (argc != 3) {
        printf("Usage: %s <file_in> <file_out>\n", argv[0]);
        exit(1);
    }

    t_image   imageIn,   imageOut;
    t_image_f gsImageIn;

    if(readPngFile(argv[1], &imageIn)) {
        printf("Error while reading image\n");
        exit(1);
    }
    rgb2gs(imageIn, &gsImageIn);

    imageOut.width    = imageIn.width;
    imageOut.height   = imageIn.height;
    mallocImage(&imageOut);
    freeImage(imageIn);

    unsigned char *gsImageInF = (unsigned char*)malloc(sizeof(unsigned char) * gsImageIn.width * gsImageIn.height);
    for(int y=0; y<gsImageIn.height; ++y) {
        for(int x=0; x<gsImageIn.width; ++x) {
            gsImageInF[y*gsImageIn.width + x] = (unsigned char)gsImageIn.data[y][x];
        }
    }

    unsigned char *kernelIn;
    unsigned char *kernelOut;
    hipMalloc((void**)&kernelIn,  gsImageIn.width * gsImageIn.height * sizeof(unsigned char));
    hipMalloc((void**)&kernelOut, imageOut.width  * imageOut.height  * sizeof(unsigned char));
    hipMemcpy(kernelIn, gsImageInF, gsImageIn.width * gsImageIn.height * sizeof(*kernelIn), hipMemcpyHostToDevice);
    freeImageF(gsImageIn);

    dim3 thrBlock(TBW, TBH);
    dim3 thrGrid(imageOut.width/TBW, imageOut.height/TBH);

    printf("Times for %d iterations:\n", RUNS);
    hipDeviceSynchronize();
    clock_t start = clock();

    for(int i=0; i<RUNS; ++i) {
        filterSobelCuda<<<thrGrid, thrBlock>>>(kernelIn, kernelOut, imageOut.width);
    }

    hipDeviceSynchronize();
    clock_t end = clock();
    float runTime = (float)(end - start) / CLOCKS_PER_SEC; // Runtime in seconds
    float numberOfMegaPixels = (imageOut.width * imageOut.height) / (1000.0f * 1000.0f);
    printf("%8.3f MegaPixels/sec (%8.3f ms, %8.3f fps)\n",
            numberOfMegaPixels*RUNS/runTime,
            runTime*1000,
            RUNS/runTime
       );

    unsigned char *gsImageOut = (unsigned char*)malloc(sizeof(unsigned char) * imageOut.width  * imageOut.height);
    hipMemcpy(gsImageOut, kernelOut, imageOut.width * imageOut.height * sizeof(*kernelOut), hipMemcpyDeviceToHost);
    for(int y=0; y<imageOut.height; ++y) {
        for(int x=0; x<imageOut.width; ++x) {
            imageOut.data[y][x].r = gsImageOut[y*imageOut.width + x];
            imageOut.data[y][x].g = gsImageOut[y*imageOut.width + x];
            imageOut.data[y][x].b = gsImageOut[y*imageOut.width + x];
        }
    }

    if(writePngFile(argv[2], imageOut)) {
        printf("Error while saving image\n");
        exit(1);
    }

    hipFree(kernelIn);
    hipFree(kernelOut);
    freeImage(imageOut);
    free(gsImageOut);
    free(gsImageInF);

    return 0;
}
